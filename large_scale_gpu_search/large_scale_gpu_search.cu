#include "hip/hip_runtime.h"
// Copyright 2021 karlluo. All rights reserved.
//
// Author: karlluo
#include <hip/hip_runtime.h>
#include <hiprand.h>

#include <cstdint>
#include <iostream>
#include <random>
#include <type_traits>
#include <ctime>

#include <thrust/device_vector.h>
#include <thrust/fill.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/random.h>
#include <thrust/transform.h>

#include <common/common_utils.hpp>

#define BLOCKSIZE 32

template <typename T>
void InitInputs(const size_t data_numbers, const size_t keys_numbers,
                thrust::host_vector<T> &h_inputs_data,
                thrust::host_vector<T> &h_keys,
                thrust::device_vector<T> &d_inputs_data,
                thrust::device_vector<T> &d_keys) {
  hiprandGenerator_t curand_gen_handler;
  // Generating random uint64_t for search
  COMMON_CURAND_CHECK(
      hiprandCreateGenerator(&curand_gen_handler, HIPRAND_RNG_QUASI_SOBOL64));
  hiprandSetGeneratorOffset (curand_gen_handler, std::time(static_cast<int>(keys_numbers)));
  hiprandSetQuasiRandomGeneratorDimensions(curand_gen_handler, keys_numbers);
  COMMON_CURAND_CHECK(hiprandGenerateLongLong(
      curand_gen_handler,
      reinterpret_cast<unsigned long long *>(
          thrust::raw_pointer_cast(d_inputs_data.data())),
      data_numbers));
  COMMON_CUDA_CHECK(hipDeviceSynchronize());
}

// cache for boundary keys indexed by threadId shared int cache[BLOCKSIZE+2] ;
// index to subset for current iteration shared int range offset;

__shared__ int cache[BLOCKSIZE + 2];

int main(int argc, char *argv[]) {
  // 500MB numbers need for search
  size_t DATA_NUMBERS = 500 * 1024 * 1024;
  // 10MB numbers keys for search
  size_t KEYS_NUMBERS = 10 * 1024 * 1024;

  thrust::host_vector<uint64_t> h_inputs_data(DATA_NUMBERS);
  thrust::host_vector<uint64_t> h_keys(KEYS_NUMBERS);
  thrust::device_vector<uint64_t> d_inputs_data(h_inputs_data);
  thrust::device_vector<uint64_t> d_keys(h_keys);
  // init random numbers for demo
  InitInputs<uint64_t>(DATA_NUMBERS, KEYS_NUMBERS, h_inputs_data, h_keys,
                       d_inputs_data, d_keys);
}